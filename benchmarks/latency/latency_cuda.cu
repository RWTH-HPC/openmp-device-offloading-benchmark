#include <cstdio>
#include <cfloat>

#include <hip/hip_runtime.h>
#include <omp.h>

#ifndef REPS
#define REPS 100000
#endif

__global__ void empty() {
    // do nothing!
}

int main(int argc, char const * argv[]) {
    int ncores;
    int ndev;
    double ** latency = NULL;
    double min_latency = DBL_MAX;
    const double usec = 1000.0 * 1000.0;

    // Determine number of cores and devices.
    hipGetDeviceCount(&ndev);
    ncores = omp_get_num_procs();

    fprintf(stdout, "---------------------------------------------------------------\n");
    fprintf(stdout, "number of cores:   %d\n", ncores);
    fprintf(stdout, "number of devices: %d\n", ndev);
    fprintf(stdout, "number of repetitions: %d\n", REPS);
    fprintf(stdout, "---------------------------------------------------------------\n");

    // Allocate the memory to store the result data.
    latency = (double **)malloc(ncores * sizeof(double *));
    for (int c = 0; c < ncores; c++) {
        latency[c] = (double *)malloc(ndev * sizeof(double));
    }

    // Print the OpenMP thread affinity info.
    /*#pragma omp parallel num_threads(ncores)
    {
        omp_display_affinity(NULL);
    }*/
    fprintf(stdout, "---------------------------------------------------------------\n");

    // Perform some warm-up to make sure that all threads are up and running,
    // and the GPUs have been properly initialized.
    fprintf(stdout, "warm up...\n");
    #pragma omp parallel num_threads(ncores)
    {
        for (int c = 0; c < ncores; c++) {
            if (omp_get_thread_num() == c) {
                for (int d = 0; d < ndev; d++) {
                    hipSetDevice(d);
                    empty<<<1,1>>>();
                    hipDeviceSynchronize();
                }
            }
            #pragma omp barrier
        }
    }
    fprintf(stdout, "---------------------------------------------------------------\n");

    // Perform the actual measurements.
    fprintf(stdout, "measurements...\n");
    double val = 0;
    #pragma omp parallel num_threads(ncores)
    {
        for (int c = 0; c < ncores; c++) {
            if (omp_get_thread_num() == c) {
                for (int d = 0; d < ndev; d++) {
                    fprintf(stdout, "running for thread=%3d and device=%2d\n", c, d);
                    fflush(stdout);
                    hipSetDevice(d);

                    double ts = omp_get_wtime();
                    for (int r = 0; r < REPS; r++) {
                        empty<<<1,1>>>();
                        hipDeviceSynchronize();
                    }
                    double te = omp_get_wtime();
                    latency[c][d] = (te - ts) / ((double) REPS) * usec;
                    if(latency[c][d] < min_latency) {
                        min_latency = latency[c][d];
                    }
                }
            }
            #pragma omp barrier
        }
    }
    fprintf(stdout, "dummy=%f\n", val);
    fprintf(stdout, "---------------------------------------------------------------\n");


    fprintf(stdout, "---------------------------------------------------------------\n");
    fprintf(stdout, "Absolute measurements (us)\n");
    fprintf(stdout, "---------------------------------------------------------------\n");
    fprintf(stdout, ";");
    for (int c = 0; c < ncores; c++) {
        fprintf(stdout, "Core %d%c", c, c<ncores-1 ? ';' : '\n');
    }
    for (int d = 0; d < ndev; d++) {
        fprintf(stdout, "GPU %d;", d);
        for (int c = 0; c < ncores; c++) {
            fprintf(stdout, "%lf%c", latency[c][d], c<ncores-1 ? ';' : '\n');
        }
    }

    fprintf(stdout, "---------------------------------------------------------------\n");
    fprintf(stdout, "Relative measurements to minimum latency\n");
    fprintf(stdout, "---------------------------------------------------------------\n");
    fprintf(stdout, ";");
    for (int c = 0; c < ncores; c++) {
        fprintf(stdout, "Core %d%c", c, c<ncores-1 ? ';' : '\n');
    }
    for (int d = 0; d < ndev; d++) {
        fprintf(stdout, "GPU %d;", d);
        for (int c = 0; c < ncores; c++) {
            fprintf(stdout, "%lf%c", (latency[c][d] / min_latency), c<ncores-1 ? ';' : '\n');
        }
    }

    return 0;
}