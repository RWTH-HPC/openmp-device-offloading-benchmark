#include "hip/hip_runtime.h"
#include "../common/util.h"
#include <cfloat>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <omp.h>

#ifndef REPS
#define REPS 100000
#endif

__global__ void empty()
{
    // do nothing!
}

int main(int argc, char *argv[])
{
    int ncores;
    int ndev;
    double *latency = NULL;
    double *latency_pp = NULL;
    double local_min_latency = DBL_MAX;
    double global_min_latency = DBL_MAX;
    const double usec = 1000.0 * 1000.0;

    MPI_Init(&argc, &argv);
    int rank, world_size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    // Determine number of cores and devices.
    hipGetDeviceCount(&ndev);
    ncores = omp_get_num_procs();

    // get representative data to fill device
    int max_threads_per_block = 0;
    int max_threads_per_mp = 0;
    int mp_count = 0;
    hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, 0);
    hipDeviceGetAttribute(&max_threads_per_mp, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
    hipDeviceGetAttribute(&mp_count, hipDeviceAttributeMultiprocessorCount, 0);
    int n_blocks_to_start = (max_threads_per_mp / max_threads_per_block) * mp_count;

    // Allocate the memory to store the result data.
    latency_pp = (double *)malloc(ndev * sizeof(double));

    print_separator(rank);

    if (rank == 0)
    {
        fprintf(stderr, "number of cores for process: %d\n", ncores);
        fprintf(stderr, "number of processes: %d\n", world_size);
        fprintf(stderr, "number of devices: %d\n", ndev);
        fprintf(stderr, "number of repetitions: %d\n", REPS);
        fprintf(stderr, "mp_count: %d\n", mp_count);
        fprintf(stderr, "max_threads_per_block: %d\n", max_threads_per_block);
        fprintf(stderr, "max_threads_per_mp: %d\n", max_threads_per_mp);
        fprintf(stderr, "n_blocks_to_start: %d\n", n_blocks_to_start);
    }

    print_separator(rank);
    print_cpu_affinity(world_size, rank);
    print_separator(rank);

    // Perform some warm-up to make sure that all threads are up and running,
    // and the GPUs have been properly initialized.
    if (rank == 0)
    {
        fprintf(stderr, "warm up...\n");
    }
    MPI_Barrier(MPI_COMM_WORLD);

    for (int c = 0; c < world_size; c++)
    {
        if (rank == c)
        {
            for (int d = 0; d < ndev; d++)
            {
                hipSetDevice(d);
                empty<<<n_blocks_to_start, max_threads_per_block>>>();
                hipDeviceSynchronize();
            }
        }
        MPI_Barrier(MPI_COMM_WORLD);
    }

    print_separator(rank);

    // Perform the actual measurements.
    if (rank == 0)
    {
        fprintf(stderr, "measurements...\n");
    }
    MPI_Barrier(MPI_COMM_WORLD);

    for (int c = 0; c < world_size; c++)
    {
        if (rank == c)
        {
            for (int d = 0; d < ndev; d++)
            {
                fprintf(stderr, "running for process=%3d and device=%2d --> ", c, d);
                hipSetDevice(d);

                double ts = omp_get_wtime();
                for (int r = 0; r < REPS; r++)
                {
                    empty<<<n_blocks_to_start, max_threads_per_block>>>();
                    hipDeviceSynchronize();
                }
                double te = omp_get_wtime();
                latency_pp[d] = (te - ts) / ((double)REPS) * usec;
                if (latency_pp[d] < local_min_latency)
                {
                    local_min_latency = latency_pp[d];
                }
                fprintf(stderr, "avg. lat = %f\n", latency_pp[d]);
            }
        }
        MPI_Barrier(MPI_COMM_WORLD);
    }

    print_separator(rank);

    if (rank == 0)
    {
        latency = (double *)malloc(world_size * ndev * sizeof(double));
    }

    MPI_Gather(latency_pp, ndev, MPI_DOUBLE, latency, ndev, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Reduce(&local_min_latency, &global_min_latency, 1, MPI_DOUBLE, MPI_MIN, 0, MPI_COMM_WORLD);

    if (rank == 0)
    {
        print_results(world_size, ndev, latency, global_min_latency, 1);
        free(latency);
    }

    free(latency_pp);

    MPI_Finalize();

    return 0;
}