#include <cstdio>
#include <cfloat>

#include <hip/hip_runtime.h>
#include <omp.h>

#ifndef REPS
#define REPS 100000
#endif

__global__ void empty() {
    // do nothing!
}

int main(int argc, char const * argv[]) {
    int ncores;
    int ndev;
    double ** latency = NULL;
    double min_latency = DBL_MAX;
    const double usec = 1000.0 * 1000.0;

    // Determine number of cores and devices.
    hipGetDeviceCount(&ndev);
    ncores = omp_get_num_procs();

    // get representative data to fill device
    int max_threads_per_block = 0;
    int max_threads_per_mp = 0;
    int mp_count = 0;
    hipDeviceGetAttribute (&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, 0);
    hipDeviceGetAttribute (&max_threads_per_mp, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
    hipDeviceGetAttribute (&mp_count, hipDeviceAttributeMultiprocessorCount, 0);
    int n_blocks_to_start = (max_threads_per_mp / max_threads_per_block) * mp_count;

    fprintf(stdout, "---------------------------------------------------------------\n");
    fprintf(stdout, "number of cores:   %d\n", ncores);
    fprintf(stdout, "number of devices: %d\n", ndev);
    fprintf(stdout, "number of repetitions: %d\n", REPS);
    fprintf(stdout, "---------------------------------------------------------------\n");
    fprintf(stdout, "mp_count: %d\n", mp_count);
    fprintf(stdout, "max_threads_per_block: %d\n", max_threads_per_block);
    fprintf(stdout, "max_threads_per_mp: %d\n", max_threads_per_mp);
    fprintf(stdout, "n_blocks_to_start: %d\n", n_blocks_to_start);
    fprintf(stdout, "---------------------------------------------------------------\n");

    // Allocate the memory to store the result data.
    latency = (double **)malloc(ncores * sizeof(double *));
    for (int c = 0; c < ncores; c++) {
        latency[c] = (double *)malloc(ndev * sizeof(double));
    }

    // Print the OpenMP thread affinity info.
    /*#pragma omp parallel num_threads(ncores)
    {
        omp_display_affinity(NULL);
    }*/
    fprintf(stdout, "---------------------------------------------------------------\n");

    // Perform some warm-up to make sure that all threads are up and running,
    // and the GPUs have been properly initialized.
    fprintf(stdout, "warm up...\n");
    #pragma omp parallel num_threads(ncores)
    {
        for (int c = 0; c < ncores; c++) {
            if (omp_get_thread_num() == c) {
                for (int d = 0; d < ndev; d++) {
                    hipSetDevice(d);
                    empty<<<n_blocks_to_start, max_threads_per_block>>>();
                    hipDeviceSynchronize();
                }
            }
            #pragma omp barrier
        }
    }
    fprintf(stdout, "---------------------------------------------------------------\n");

    // Perform the actual measurements.
    fprintf(stdout, "measurements...\n");
    double val = 0;
    #pragma omp parallel num_threads(ncores)
    {
        for (int c = 0; c < ncores; c++) {
            if (omp_get_thread_num() == c) {
                for (int d = 0; d < ndev; d++) {
                    fprintf(stdout, "running for thread=%3d and device=%2d\n", c, d);
                    fflush(stdout);
                    hipSetDevice(d);

                    double ts = omp_get_wtime();
                    for (int r = 0; r < REPS; r++) {
                        empty<<<n_blocks_to_start, max_threads_per_block>>>();
                        hipDeviceSynchronize();
                    }
                    double te = omp_get_wtime();
                    latency[c][d] = (te - ts) / ((double) REPS) * usec;
                    if(latency[c][d] < min_latency) {
                        min_latency = latency[c][d];
                    }
                }
            }
            #pragma omp barrier
        }
    }
    fprintf(stdout, "dummy=%f\n", val);
    fprintf(stdout, "---------------------------------------------------------------\n");


    fprintf(stdout, "---------------------------------------------------------------\n");
    fprintf(stdout, "Absolute measurements (us)\n");
    fprintf(stdout, "---------------------------------------------------------------\n");
    fprintf(stdout, ";");
    for (int c = 0; c < ncores; c++) {
        fprintf(stdout, "Core %d%c", c, c<ncores-1 ? ';' : '\n');
    }
    for (int d = 0; d < ndev; d++) {
        fprintf(stdout, "GPU %d;", d);
        for (int c = 0; c < ncores; c++) {
            fprintf(stdout, "%lf%c", latency[c][d], c<ncores-1 ? ';' : '\n');
        }
    }

    fprintf(stdout, "---------------------------------------------------------------\n");
    fprintf(stdout, "Relative measurements to minimum latency\n");
    fprintf(stdout, "---------------------------------------------------------------\n");
    fprintf(stdout, ";");
    for (int c = 0; c < ncores; c++) {
        fprintf(stdout, "Core %d%c", c, c<ncores-1 ? ';' : '\n');
    }
    for (int d = 0; d < ndev; d++) {
        fprintf(stdout, "GPU %d;", d);
        for (int c = 0; c < ncores; c++) {
            fprintf(stdout, "%lf%c", (latency[c][d] / min_latency), c<ncores-1 ? ';' : '\n');
        }
    }

    return 0;
}