#include <cstdio>
#include <cstring>
#include <cfloat>

#include <hip/hip_runtime.h>
#include <omp.h>

#ifndef REPS
#define REPS 10
#endif

#ifndef INCLUDE_ALLOC
#define INCLUDE_ALLOC 1
#endif

__global__ void empty(size_t n, char * array) {
    // do nothing!
}

int main(int argc, char const * argv[]) {
    int ncores;
    int ndev;
    double *** bandwidth = NULL;
    double * min_bandwidth = NULL;

    const int nsizes = 3;
    size_t array_sizes_bytes[3] = {10000000, 100000000, 1000000000};
    const size_t MAX_BUF_SIZE = 1000000000;

    // Determine number of cores and devices.
    hipGetDeviceCount(&ndev);
    ncores = omp_get_num_procs();

    // get representative data to fill device
    int max_threads_per_block = 0;
    int max_threads_per_mp = 0;
    int mp_count = 0;
    hipDeviceGetAttribute (&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, 0);
    hipDeviceGetAttribute (&max_threads_per_mp, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
    hipDeviceGetAttribute (&mp_count, hipDeviceAttributeMultiprocessorCount, 0);
    int n_blocks_to_start = (max_threads_per_mp / max_threads_per_block) * mp_count;

    fprintf(stdout, "---------------------------------------------------------------\n");
    fprintf(stdout, "number of array sizes: %d\n", nsizes);
    fprintf(stdout, "number of cores:   %d\n", ncores);
    fprintf(stdout, "number of devices: %d\n", ndev);
    fprintf(stdout, "number of repetitions: %d\n", REPS);
    fprintf(stdout, "---------------------------------------------------------------\n");
    fprintf(stdout, "mp_count: %d\n", mp_count);
    fprintf(stdout, "max_threads_per_block: %d\n", max_threads_per_block);
    fprintf(stdout, "max_threads_per_mp: %d\n", max_threads_per_mp);
    fprintf(stdout, "n_blocks_to_start: %d\n", n_blocks_to_start);
    fprintf(stdout, "---------------------------------------------------------------\n");

    // Allocate the memory to store the result data.
    bandwidth = (double ***)malloc(nsizes * sizeof(double **));
    min_bandwidth = (double *)malloc(nsizes * sizeof(double));
    for (int s = 0; s < nsizes; s++) {
        bandwidth[s] = (double **)malloc(ncores * sizeof(double *));
        min_bandwidth[s] = DBL_MAX;
        for (int c = 0; c < ncores; c++) {
            bandwidth[s][c] = (double *)malloc(ndev * sizeof(double));
        }
    }

    /*// Print the OpenMP thread affinity info.
    #pragma omp parallel num_threads(ncores)
    {
        omp_display_affinity(NULL);
    }*/
    
    // Allocate per thread buffers
    char ** per_thread_buffs = (char **)malloc(ncores * sizeof(char *));
    #pragma omp parallel num_threads(ncores)
    {
        int cur_thread = omp_get_thread_num();
        per_thread_buffs[cur_thread] = (char *)malloc(MAX_BUF_SIZE);
        // init buffer using first-touch
        memset(per_thread_buffs[cur_thread], 0, MAX_BUF_SIZE);
    }

    fprintf(stdout, "---------------------------------------------------------------\n");

    // Perform some warm-up to make sure that all threads are up and running,
    // and the GPUs have been properly initialized.
    fprintf(stdout, "warm up...\n");
    #pragma omp parallel num_threads(ncores)
    {
        for (int c = 0; c < ncores; c++) {
            if (omp_get_thread_num() == c) {
                for (int d = 0; d < ndev; d++) {
                    hipSetDevice(d);
                    empty<<<n_blocks_to_start, max_threads_per_block>>>(d, NULL);
                    hipDeviceSynchronize();
                }
            }
            #pragma omp barrier
        }
    }
    fprintf(stdout, "---------------------------------------------------------------\n");

    // Perform the actual measurements.
    fprintf(stdout, "measurements...\n");
    #pragma omp parallel num_threads(ncores)
    {
        int cur_thread = omp_get_thread_num();
        for (int c = 0; c < ncores; c++) {
            if (cur_thread == c) {
                for (int s = 0; s < nsizes; s++) {
                    size_t cur_size     = array_sizes_bytes[s];
                    double tmp_size_mb  = ((double)cur_size / 1e6);

                    for (int d = 0; d < ndev; d++) {
                        fprintf(stdout, "running for thread=%3d, size=%7.2fMB and device=%2d\n", c, tmp_size_mb, d);
                        fflush(stdout);
                        hipSetDevice(d);
                        
                        // allocate and initialize data once (first-touch)
                        // char * buffer = (char *)malloc(cur_size);
                        // memset(buffer, 0, cur_size);
                        char * buffer_dev = NULL;
                        char * buffer = per_thread_buffs[cur_thread];
#if !INCLUDE_ALLOC
                        cudaMalloc(&buffer_dev, sizeof(*buffer_dev) * cur_size);
#endif
                        double ts = omp_get_wtime();
                        for (int r = 0; r < REPS; r++) {
#if INCLUDE_ALLOC
                            hipMalloc(&buffer_dev, sizeof(*buffer_dev) * cur_size);
#endif
                            hipMemcpy(buffer_dev, buffer, sizeof(*buffer) * cur_size, hipMemcpyHostToDevice);
                            empty<<<n_blocks_to_start, max_threads_per_block>>>(cur_size, buffer_dev);
                            hipMemcpy(buffer, buffer_dev, sizeof(*buffer) * cur_size, hipMemcpyDeviceToHost);
                            hipDeviceSynchronize();
#if INCLUDE_ALLOC
                            hipFree(buffer_dev);
#endif
                        }
                        double te = omp_get_wtime();
                        double avg_time_sec = (te - ts) / ((double) REPS);
                        bandwidth[s][c][d] = tmp_size_mb * 2 / avg_time_sec;
                        if(bandwidth[s][c][d] < min_bandwidth[s]) {
                            min_bandwidth[s] = bandwidth[s][c][d];
                        }

#if !INCLUDE_ALLOC
                        cudaFree(buffer_dev);
#endif
                    }
                }
            }
            #pragma omp barrier
        }
    }
    fprintf(stdout, "---------------------------------------------------------------\n");

    // free memory and cleanup
    for(int i = 0; i < ncores; i++) {
        free(per_thread_buffs[i]);
    }
    free(per_thread_buffs);

    fprintf(stdout, "---------------------------------------------------------------\n");
    fprintf(stdout, "Absolute measurements (MB/s)\n");
    fprintf(stdout, "---------------------------------------------------------------\n");
    for (int s = 0; s < nsizes; s++) {
        size_t cur_size = array_sizes_bytes[s];
        fprintf(stdout, "##### Problem Size: %.2f KB\n", cur_size / 1000.0);
        fprintf(stdout, ";");
        for (int c = 0; c < ncores; c++) {
            fprintf(stdout, "Core %d%c", c, c<ncores-1 ? ';' : '\n');
        }
        for (int d = 0; d < ndev; d++) {
            fprintf(stdout, "GPU %d;", d);
            for (int c = 0; c < ncores; c++) {
                fprintf(stdout, "%lf%c", bandwidth[s][c][d], c<ncores-1 ? ';' : '\n');
            }
        }
    }
    fprintf(stdout, "\n\n");
    for (int c = 0; c < ncores; c++) {
        fprintf(stdout, "##### Core: %d\n", c);
        fprintf(stdout, ";");
        for (int s = 0; s < nsizes; s++) {
            size_t cur_size = array_sizes_bytes[s];
            fprintf(stdout, "%.2f KB%c", cur_size / 1000.0, s<nsizes-1 ? ';' : '\n');
        }
        for (int d = 0; d < ndev; d++) {
            fprintf(stdout, "GPU %d;", d);
            for (int s = 0; s < nsizes; s++) {
                fprintf(stdout, "%lf%c", bandwidth[s][c][d], s<nsizes-1 ? ';' : '\n');
            }
        }
    }

    fprintf(stdout, "---------------------------------------------------------------\n");
    fprintf(stdout, "Relative measurements to minimum bandwidth for size\n");
    fprintf(stdout, "---------------------------------------------------------------\n");
    for (int s = 0; s < nsizes; s++) {
        size_t cur_size = array_sizes_bytes[s];
        fprintf(stdout, "##### Problem Size: %.2f KB\n", cur_size / 1000.0);
        fprintf(stdout, ";");
        for (int c = 0; c < ncores; c++) {
            fprintf(stdout, "Core %d%c", c, c<ncores-1 ? ';' : '\n');
        }
        for (int d = 0; d < ndev; d++) {
            fprintf(stdout, "GPU %d;", d);
            for (int c = 0; c < ncores; c++) {
                fprintf(stdout, "%lf%c", bandwidth[s][c][d] / min_bandwidth[s], c<ncores-1 ? ';' : '\n');
            }
        }
    }
    fprintf(stdout, "\n\n");
    for (int c = 0; c < ncores; c++) {
        fprintf(stdout, "##### Core: %d\n", c);
        fprintf(stdout, ";");
        for (int s = 0; s < nsizes; s++) {
            size_t cur_size = array_sizes_bytes[s];
            fprintf(stdout, "%.2f KB%c", cur_size / 1000.0, s<nsizes-1 ? ';' : '\n');
        }
        for (int d = 0; d < ndev; d++) {
            fprintf(stdout, "GPU %d;", d);
            for (int s = 0; s < nsizes; s++) {
                fprintf(stdout, "%lf%c", bandwidth[s][c][d] / min_bandwidth[s], s<nsizes-1 ? ';' : '\n');
            }
        }
    }

    return 0;
}